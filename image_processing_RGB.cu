
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <png.h>
#include <sys/time.h>

typedef struct
{
    int height;
    int width;
    int pixel_size;
    png_infop info_ptr;
    png_byte *buf;
} PNG_RAW;

long long timeInMilliseconds(void)
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (((long long)tv.tv_sec) * 1000) + (tv.tv_usec / 1000);
}

PNG_RAW *read_png(char *file_name)
{
    PNG_RAW *png_raw = (PNG_RAW *)malloc(sizeof(PNG_RAW));
    FILE *fp = fopen(file_name, "rb");
    png_structp png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_read_png(png_ptr, info_ptr, PNG_TRANSFORM_IDENTITY, NULL);
    png_bytepp row_pointers = png_get_rows(png_ptr, info_ptr);
    int width = png_get_image_width(png_ptr, info_ptr);
    int height = png_get_image_height(png_ptr, info_ptr);
    int pixel_size = 3;
    png_raw->width = width;
    png_raw->height = height;
    png_raw->pixel_size = pixel_size;
    png_raw->buf = (png_byte *)malloc(width * height * pixel_size * sizeof(png_byte));
    png_raw->info_ptr = info_ptr;
    int k = 0;
    for (int i = 0; i < height; i++)
        for (int j = 0; j < width * pixel_size; j++)
        {
            png_raw->buf[k++] = row_pointers[i][j];
        }
    png_destroy_read_struct(&png_ptr, NULL, NULL);
    fclose(fp);
    return png_raw;
}

void write_png(char *file_name, PNG_RAW *png_raw)
{
    FILE *fp = fopen(file_name, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_init_io(png_ptr, fp);
    png_infop info_ptr = png_raw->info_ptr;
    int width = png_raw->width;
    int height = png_raw->height;
    int pixel_size = png_raw->pixel_size;
    png_bytepp row_pointers;
    row_pointers = (png_bytepp)malloc(height * sizeof(png_bytep));
    for (int i = 0; i < height; i++)
        row_pointers[i] = (png_bytep)malloc(width * pixel_size);
    int k = 0;
    for (int i = 0; i < height; i++)
        for (int j = 0; j < width * pixel_size; j++)
        {
            row_pointers[i][j] = png_raw->buf[k++];
        }

    png_set_rows(png_ptr, info_ptr, row_pointers);
    png_write_png(png_ptr, info_ptr, PNG_TRANSFORM_IDENTITY, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    for (int i = 0; i < height; i++)
        free(row_pointers[i]);
    free(row_pointers);
    fclose(fp);
}

void process_on_host(PNG_RAW *png_raw)
{
    long long start = timeInMilliseconds();
    for (int i = 0; i < png_raw->width * png_raw->height; i++)
    {
        int index = i * png_raw->pixel_size;
        png_raw->buf[index] = (png_byte)255; // Red channel
        png_raw->buf[index + 1] = (png_byte)0; // Green channel
        png_raw->buf[index + 2] = (png_byte)0; // Blue channel
    }
    long long end = timeInMilliseconds();
    printf("Timing on host is %lld millis\n", end - start);
}

__global__ void BlurKernel(png_byte *d_P, int height, int width, int pixel_size)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int index = (Row * width + Col) * pixel_size;

    if (Row < height && Col < width)
    {
        int sumR = 0, sumG = 0, sumB = 0;
        int count = 0;

        for (int i = -2; i <= 2; i++)
        {
            for (int j = -2; j <= 2; j++)
            {
                int curRow = Row + i;
                int curCol = Col + j;

                if (curRow >= 0 && curRow < height && curCol >= 0 && curCol < width)
                {
                    int curIndex = (curRow * width + curCol) * pixel_size;
                    sumR += d_P[curIndex];
                    sumG += d_P[curIndex + 1];
                    sumB += d_P[curIndex + 2];
                    count++;
                }
            }
        }
        png_byte avgR = sumR / count;
        png_byte avgG = sumG / count;
        png_byte avgB = sumB / count;

        d_P[index] = avgR;
        d_P[index + 1] = avgG;
        d_P[index + 2] = avgB;
    }
}

void process_blurring_on_device(PNG_RAW *png_raw)
{
    int m = png_raw->height;
    int n = png_raw->width;
    int pixel_size = png_raw->pixel_size;

    png_byte *d_P;
    hipError_t err;

    long long start = timeInMilliseconds();

    err = hipMalloc((void **)&d_P, m * n * pixel_size * sizeof(png_byte));
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    hipMemcpy(d_P, png_raw->buf, m * n * pixel_size, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (m + blockDim.y - 1) / blockDim.y);

    BlurKernel<<<gridDim, blockDim>>>(d_P, m, n, pixel_size);

    hipMemcpy(png_raw->buf, d_P, m * n * pixel_size * sizeof(png_byte), hipMemcpyDeviceToHost);

    long long end = timeInMilliseconds();

    printf("\n");
    printf("Blurring your image... \n");
    printf("Timing on Device is %lld millis\n", end - start);

    hipFree(d_P);
}

__global__ void EdgeDetectionKernel(png_byte *d_P, int height, int width, int pixel_size)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int index = (Row * width + Col) * pixel_size;

    if (Row < height && Col < width)
    {
        int GxR = 0, GxG = 0, GxB = 0;
        int GyR = 0, GyG = 0, GyB = 0;

        int sobelMaskX[3][3] = { { -1, 0, 1 },
                                 { -2, 0, 2 },
                                 { -1, 0, 1 } };

        int sobelMaskY[3][3] = { { -1, -2, -1 },
                                 { 0, 0, 0 },
                                 { 1, 2, 1 } };

        for (int i = -1; i <= 1; i++)
        {
            for (int j = -1; j <= 1; j++)
            {
                int curRow = Row + i;
                int curCol = Col + j;

                if (curRow >= 0 && curRow < height && curCol >= 0 && curCol < width)
                {
                    int curIndex = (curRow * width + curCol) * pixel_size;
                    int maskValueX = sobelMaskX[i + 1][j + 1];
                    int maskValueY = sobelMaskY[i + 1][j + 1];

                    GxR += d_P[curIndex] * maskValueX;
                    GxG += d_P[curIndex + 1] * maskValueX;
                    GxB += d_P[curIndex + 2] * maskValueX;

                    GyR += d_P[curIndex] * maskValueY;
                    GyG += d_P[curIndex + 1] * maskValueY;
                    GyB += d_P[curIndex + 2] * maskValueY;
                }
            }
        }

        int gradientMagnitudeR = sqrtf(GxR * GxR + GyR * GyR);
        int gradientMagnitudeG = sqrtf(GxG * GxG + GyG * GyG);
        int gradientMagnitudeB = sqrtf(GxB * GxB + GyB * GyB);

        png_byte normalizedMagnitudeR = (png_byte)(gradientMagnitudeR / 255.0f * 255.0f);
        png_byte normalizedMagnitudeG = (png_byte)(gradientMagnitudeG / 255.0f * 255.0f);
        png_byte normalizedMagnitudeB = (png_byte)(gradientMagnitudeB / 255.0f * 255.0f);

        d_P[index] = normalizedMagnitudeR;
        d_P[index + 1] = normalizedMagnitudeG;
        d_P[index + 2] = normalizedMagnitudeB;
    }
}

void process_edge_detection_on_device(PNG_RAW *png_raw)
{
    int m = png_raw->height;
    int n = png_raw->width;
    int pixel_size = png_raw->pixel_size;

    png_byte *d_P;
    hipError_t err;

    long long start = timeInMilliseconds();

    err = hipMalloc((void **)&d_P, m * n * pixel_size * sizeof(png_byte));
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    hipMemcpy(d_P, png_raw->buf, m * n * pixel_size, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (m + blockDim.y - 1) / blockDim.y);

    EdgeDetectionKernel<<<gridDim, blockDim>>>(d_P, m, n, pixel_size);

    hipMemcpy(png_raw->buf, d_P, m * n * pixel_size * sizeof(png_byte), hipMemcpyDeviceToHost);

    long long end = timeInMilliseconds();

    printf("\n");
    printf("Detecting edges... \n");
    printf("Timing on Device is %lld millis\n", end - start);

    hipFree(d_P);
}

__global__ void SharpeningKernel(png_byte *d_P, int height, int width, int pixel_size)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int index = (row * width + col) * pixel_size;

    if (row < height && col < width)
    {
        float center = d_P[index];
        float left = (col > 0) ? d_P[index - pixel_size] : 0.0f;
        float right = (col < width - 1) ? d_P[index + pixel_size] : 0.0f;
        float top = (row > 0) ? d_P[index - width * pixel_size] : 0.0f;
        float bottom = (row < height - 1) ? d_P[index + width * pixel_size] : 0.0f;

        float sharpened = 5.0f * center - (left + right + top + bottom);

        sharpened = fminf(fmaxf(sharpened, 0.0f), 255.0f);

        d_P[index] = (png_byte)sharpened;
    }
}

void process_sharpening_on_device(PNG_RAW *png_raw)
{
    int m = png_raw->height;
    int n = png_raw->width;
    int pixel_size = png_raw->pixel_size;

    png_byte *d_P;
    hipError_t err;

    long long start = timeInMilliseconds();

    err = hipMalloc((void **)&d_P, m * n * pixel_size * sizeof(png_byte));
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    hipMemcpy(d_P, png_raw->buf, m * n * pixel_size, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (m + blockDim.y - 1) / blockDim.y);

    SharpeningKernel<<<gridDim, blockDim>>>(d_P, m, n, pixel_size);

    hipMemcpy(png_raw->buf, d_P, m * n * pixel_size * sizeof(png_byte), hipMemcpyDeviceToHost);

    long long end = timeInMilliseconds();

    printf("\n");
    printf("Sharpening your image... \n");
    printf("Timing on Device is %lld millis\n", end - start);

    hipFree(d_P);
}

int main(int argc, char **argv)
{
    int on_host = 0;
    int option;

    if (argv[3] != NULL && strcmp(argv[3], "-d") == 0)
        on_host = 0;

    PNG_RAW *png_raw = read_png(argv[1]);
    if (png_raw->pixel_size != 3)
    {
        printf("Error, png file must be on 3 Bytes per pixel\n");
        exit(0);
    }
    else
        printf("RGB Processing for Image of %d x %d pixels\n", png_raw->width, png_raw->height);

    if (on_host){
      process_on_host(png_raw);
    }
    else{

      printf("\n");
      printf("Choose what to do with the image: \n");
      printf("1. Blurring \n");
      printf("2. Edge Detection \n");
      printf("3. Sharpening \n");
      printf("------------------\n");
      printf("Enter your choice: ");

      scanf("%d", &option);

      if (option == 1){
        process_blurring_on_device(png_raw);
      }

      if(option == 2){
        process_edge_detection_on_device(png_raw);
      }

      if(option == 3){
        process_sharpening_on_device(png_raw);
      }

    }   

    write_png(argv[2], png_raw);

    printf("Processing finished\n");

    free(png_raw->buf);
    free(png_raw);

    return 0;
}

